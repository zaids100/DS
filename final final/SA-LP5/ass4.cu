#include "hip/hip_runtime.h"
// CUDA Program for Addition of Two Large Vectors

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);

    // Allocate host memory
    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < n; i++) {
        if (c[i] != 2 * i) {
            printf("Error: c[%d] = %d\n", i, c[i]);
            break;
        }
    }

    printf("Vector addition successful!\n");

    // Free memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}





// CUDA Program for Matrix Multiplication
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matrix_multiply(float *a, float *b, float *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int n = 1024;
    size_t size = n * n * sizeof(float);

    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate host memory
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < n * n; ++i) {
        a[i] = i % n;
        b[i] = i % n;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);

    // Launch and time the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrix_multiply<<<blocks, threads>>>(d_a, d_b, d_c, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    // Copy result to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Elapsed time for matrix multiplication: %.2f ms\n", elapsed_time);

    // Free memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}
